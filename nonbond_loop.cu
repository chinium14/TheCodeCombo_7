
#include <hip/hip_runtime.h>
#ifdef CUDA
#ifndef PRESSURE
#include "cuda.h"

__global__ void nonbond_loop(double * force, double * enonbond, double * enonbonds, int boxns, real3 * atom_xyz, double bl, double bh, interb *ljset, int k_dev){
	int a = threadIdx.x + blockIdx.x * blockDim.x;
	int b = threadIdx.y + blockIdx.y * blockDim.y;
	int id = a + b * blockDim.x * gridDim.x;
	if (b >= a){
		double fxa = 0.0;
		double fya = 0.0;
		double fza = 0.0;
		double ax = atom_xyz[a].x;
		double ay = atom_xyz[a].y;
		double az = atom_xyz[a].z;
		double bx = atom_xyz[b].x;
		double by = atom_xyz[b].y;
		double bz = atom_xyz[b].z;
		double dx = ax - bx;
		double dy = ay - by;
		double dz = az - bz;
		double pbcx = 0.0;
		double pbcy = 0.0;
		double pbcz = 0.0;
                
		if(dx >  bh) pbcx =- bl;
                if(dx < -bh) pbcx =+ bl;
                if(dy >  bh) pbcy =- bl;
                if(dy < -bh) pbcy =+ bl;
                if(dz >  bh) pbcz =- bl;
                if(dz < -bh) pbcz =+ bl;
		
		dx += pbcx;
                dy += pbcy;
                dz += pbcz;
                double dr2 = dx*dx + dy*dy + dz*dz;
                if(dr2 < rc2) {
        
//      //      printf("%d %d  %lf \n",a,b,sqrt(dr2));
		double eps = ljset[k].pot[a][b].eps;
                double sig = ljset[k].pot[a][b].sig;
                double qq  = ljset[k].pot[a][b].qq;
        
                double sr2  = (sig * sig) / dr2;
                double sr6  = sr2 * sr2 * sr2;
                double sr12 = sr6 * sr6;
        
                double sr2s  = (sig * sig) / rc2;
                double sr6s  = sr2s * sr2s * sr2s;
                double sr12s = sr6s * sr6s;
		
		force[id] = (12.0*eps/dr2) * (sr12 -sr6);
		enonbond[id] += eps * (sr12 - 2*sr6);
		enonbonds[id] += eps * (sr12 - 2*sr6);
		}else{
			force[id] = 0;
			enonbond[id] = 0;
			enonnbonds[id] = 0;
		}
		__syncthreads();
		int half = boxns/2;
		while (half != 0){
			force[id] +=force[id+half];
			__syncthreads();
			half /= 2;
		}
	}
}

void cnonbond(int ibox, double *energy)
{
	int k = ibox;
	double enonbond = 0.0;
	double enonbonds = 0.0;
	double ecoulomb = 0.0;
	
	double enonbond_temp;
	double enonbonds_temp;
	double ecoulomb_temp;
	
	int k_dev, boxns_dev;
	double *enonbond_dev;
	double *enonbonds_dev;
	double *force_dev;
	cudaMalloc((void*)&k_dev,sizeof(int));
	cudaMalloc((void*)&boxns_dev,sizeof(int));
	cudaMalloc((void**)&enonbond_dev,boxns[k].boxns*boxns[k].boxns*sizeof(double));
	cudaMalloc((void**)&enonbonds_dev,boxns[k].boxns*boxns[k].boxns*sizeof(double));
	cudaMalloc((void**)&force_dev,boxns[k].boxns*boxns[k].boxns*sizeof(double));
	cudaMemcpy(k_dev,k,sizeof(int),cudaMemcpyHostToDevice);	
	cudaMemcpy(boxns_dev,box[k].boxns,sizeof(int),cudaMemcpyHostToDevice);
	
	  double rc   = sqrt(box[k].rc2);
	  double rc2  = box[k].rc2;
	  double rfcs = (sim.epsRF[k] - 1.0) / (2.0*sim.epsRF[k] + 1.0);
	  double rfc  = rfcs / (rc2*rc);
	  double bl   = box[k].boxl;
	  double bh   = box[k].boxh;
		
?	  double bl_dev;
?	  double bh_dev;
	  cudaMemcpy(bl_dev,bl, sizeof(double),cudaMemcpyHostToDevice);
	  cudaMemcpy(bh_dev,bh, sizeof(double),cudaMemcpyHostToDevice);
//Generate atom coordinate information
	real3 * atom_xyz;
	atom_xyz = (real3 *)malloc(box[k].boxns * sizeof(real3));
	for (int i = 0; i < box[k].boxns-1; i++){
		atom_xyz[i] = make_real3(atom[k][i].x, atom[k][i].y, atom[k][i].z)
	}
	real3 * atom_xyz_dev;
	cudaMalloc((void **)&atom_xyz_dev, box[k].boxns*sizeof(real3));
	cudaMemcpy(atom_xyz_dev,atom_xyz,box[k].boxns*sizeof(real3),cudaMemcpyHostToDevice);
// Perform calculation on GPU of nonbond forces
	dim3 threads(16,16);
	dim3 blocks((boxns+15)/16,(boxns+15)/16);
	nonbond_loop<<<blocks,threads>>>(force_dev,enonbond_dev,enonbonds_dev,boxns,atom_xyz_dev, bl_dev, bh_dev, );

	cudaMemcpy(k,k_dev,sizeof(int),cudaMemcpyDeviceToHost);	
	cudaMemcpy(enonbond,enonbond_dev[0],sizeof(double),cudaMemcpyDeviceToHost);	
	cudaMemcpy(enonbonds,enonbonds_dev[0],sizeof(double),cudaMemcpyDeviceToHost);	
	cudaMemcpy(force,force_dev[0],sizeof(double),cudaMemcpyDeviceToHost);
	cudaFree(k_dev);
	cudaFree(enonbond_dev);
	cudaFree(enonbonds_dev);
	cudaFree(force_dev);
	cudaFree(atom_xyz_dev);
	free(atom_xyz);
}
#endif
#endif
